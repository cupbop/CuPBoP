
#include <hip/hip_runtime.h>
#include <cstdio>
#include <random>
#include <stdint.h>
#include <sys/time.h>

__global__ void vecadd(int *a, int *b, int *c, int n) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < n) {
    c[tid] += a[tid] + b[tid];
  }
}

int main() {
  int size = 512;
  int *h_a, *h_b, *h_c;
  h_a = (int *)malloc(size * sizeof(int));
  h_b = (int *)malloc(size * sizeof(int));
  h_c = (int *)malloc(size * sizeof(int));

  for (int i = 0; i < size; i++) {
    h_a[i] = i;
    h_b[i] = 2 * i;
    h_c[i] = 0;
  }
  int *d_a, *d_b, *d_c;
  hipMalloc(&d_a, size * sizeof(int));
  hipMalloc(&d_b, size * sizeof(int));
  hipMalloc(&d_c, size * sizeof(int));

  hipMemcpy(d_a, h_a, size * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, size * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_c, h_c, size * sizeof(int), hipMemcpyHostToDevice);

  // Launch with different block size
  vecadd<<<std::ceil(size / 16.0), 16>>>(d_a, d_b, d_c, size);
  vecadd<<<std::ceil(size / 32.0), 32>>>(d_a, d_b, d_c, size);
  vecadd<<<std::ceil(size / 42.0), 42>>>(d_a, d_b, d_c, size);

  hipMemcpy(h_c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);

  // Verify the result
  for (int i = 0; i < size; i++) {
    if (h_c[i] != 3 * (h_a[i] + h_b[i])) {
      printf("Error at index %d\n", i);
      return 1;
    }
  }
  printf("PASS\n");
  return 0;
}
