
#include <hip/hip_runtime.h>
#include <cstdio>
#include <random>
#include <stdint.h>
#include <sys/time.h>

__global__ void Histogram(uint32_t *pixels, uint32_t *histogram,
                          uint32_t num_colors, uint32_t num_pixels) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int gsize = gridDim.x * blockDim.x;

  uint32_t priv_hist[256];
  for (uint32_t i = 0; i < num_colors; i++) {
    priv_hist[i] = 0;
  }
  uint32_t index = tid;
  while (index < num_pixels) {
    uint32_t color = pixels[index];
    priv_hist[color]++;
    index += gsize;
  }

  __syncthreads();
  for (uint32_t i = 0; i < num_colors; i++) {
    if (priv_hist[i] > 0) {
      atomicAdd(histogram + i, priv_hist[i]);
    }
  }
}

__global__ void opt_Histogram(uint32_t *pixels, uint32_t *histogram,
                              uint32_t num_colors, uint32_t num_pixels) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int gsize = gridDim.x * blockDim.x;

  uint32_t priv_hist[256];
  for (uint32_t i = 0; i < num_colors; i++) {
    priv_hist[i] = 0;
  }
  uint32_t index = tid;
  __shared__ bool has_activated_thread;
  bool activated = true;
  do {
    has_activated_thread = false;
    __syncthreads();
    activated = activated & (index < num_pixels);
    has_activated_thread |= activated;
    if (activated) {
      uint32_t color = pixels[index];
      priv_hist[color]++;
      index += gsize;
    }
  } while (has_activated_thread);
  __syncthreads();
  for (uint32_t i = 0; i < num_colors; i++) {
    if (priv_hist[i] > 0) {
      atomicAdd(histogram + i, priv_hist[i]);
    }
  }
}

timeval time_start, time_end;
unsigned int totalKernelTime;

int main() {
  int num_pixel_ = 7;
  int num_color_ = 256;
  uint32_t *pixels_ = new uint32_t[num_pixel_ * 2];
  unsigned int seed = 42;
  for (uint32_t i = 0; i < num_pixel_ * 2; i++) {
    pixels_[i] = rand_r(&seed) % num_color_;
  }

  uint32_t *opt_histogram_ = new uint32_t[num_color_]();
  uint32_t *no_opt_histogram_ = new uint32_t[num_color_]();

  uint32_t *d_histogram;
  uint32_t *d_pixels;
  hipMalloc(&d_pixels, num_pixel_ * 2 * sizeof(uint32_t));
  hipMemcpy(d_pixels, pixels_, num_pixel_ * 2 * sizeof(uint32_t),
             hipMemcpyHostToDevice);
  // optimized
  hipMalloc(&d_histogram, num_color_ * sizeof(uint32_t));
  opt_Histogram<<<8192 / 64, 64>>>(d_pixels, d_histogram, num_color_,
                                   num_pixel_);
  hipDeviceSynchronize();
  hipMemcpy(opt_histogram_, d_histogram, num_color_ * sizeof(uint32_t),
             hipMemcpyDeviceToHost);

  // unoptimized
  hipMalloc(&d_histogram, num_color_ * sizeof(uint32_t));
  Histogram<<<8192 / 64, 64>>>(d_pixels, d_histogram, num_color_, num_pixel_);
  hipDeviceSynchronize();
  hipMemcpy(no_opt_histogram_, d_histogram, num_color_ * sizeof(uint32_t),
             hipMemcpyDeviceToHost);
  for (int i = 0; i < num_color_; i++) {
    if (no_opt_histogram_[i] != opt_histogram_[i]) {
      printf("Error!\n");
      printf("%d %d\n", no_opt_histogram_[i], opt_histogram_[i]);
      exit(1);
    }
  }
  printf("PASS\n");
  return 0;
}
