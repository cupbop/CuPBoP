
#include <hip/hip_runtime.h>
#include <cstdio>
#include <random>
#include <stdint.h>
#include <sys/time.h>

const int SIZE = 512;
__global__ void vecadd(int *a, int *b, int *c) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < SIZE) {
    c[tid] += a[tid] + b[tid];
  }
}

int main() {
  int *h_a, *h_b, *h_c;
  h_a = (int *)malloc(SIZE * sizeof(int));
  h_b = (int *)malloc(SIZE * sizeof(int));
  h_c = (int *)malloc(SIZE * sizeof(int));

  for (int i = 0; i < SIZE; i++) {
    h_a[i] = i;
    h_b[i] = 2 * i;
    h_c[i] = 0;
  }
  int *d_a, *d_b, *d_c;
  hipMalloc(&d_a, SIZE * sizeof(int));
  hipMalloc(&d_b, SIZE * sizeof(int));
  hipMalloc(&d_c, SIZE * sizeof(int));

  hipMemcpy(d_a, h_a, SIZE * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, SIZE * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_c, h_c, SIZE * sizeof(int), hipMemcpyHostToDevice);

  vecadd<<<SIZE/16, 16>>>(d_a, d_b, d_c);

  hipMemcpy(h_c, d_c, SIZE * sizeof(int), hipMemcpyDeviceToHost);

  // Verify the result
  for (int i = 0; i < SIZE; i++) {
    if (h_c[i] != (h_a[i] + h_b[i])) {
      printf("Error at index %d\n", i);
      return 1;
    }
  }
  printf("PASS\n");
  return 0;
}
